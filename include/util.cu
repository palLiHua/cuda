#ifndef UTIL_H
#define UTIL_H
#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
    }\
}\

#define CHECK_CUSPARSE(call)                                                   \
{                                                                              \
    hipsparseStatus_t err;                                                      \
    if ((err = (call)) != HIPSPARSE_STATUS_SUCCESS)                             \
    {                                                                          \
        fprintf(stderr, "Got error %d at %s:%d\n", err, __FILE__, __LINE__);   \
        hipError_t cuda_err = hipGetLastError();                             \
        if (cuda_err != hipSuccess)                                           \
        {                                                                      \
            fprintf(stderr, "  CUDA error \"%s\" also detected\n",             \
                    hipGetErrorString(cuda_err));                             \
        }                                                                      \
        exit(1);                                                               \
    }                                                                          \
}

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif

#ifdef _WIN32
int gettimeofday(struct timeval *tp, void *tzp)
{
  time_t clock;
  struct tm tm;
  SYSTEMTIME wtm;
  GetLocalTime(&wtm);
  tm.tm_year   = wtm.wYear - 1900;
  tm.tm_mon   = wtm.wMonth - 1;
  tm.tm_mday   = wtm.wDay;
  tm.tm_hour   = wtm.wHour;
  tm.tm_min   = wtm.wMinute;
  tm.tm_sec   = wtm.wSecond;
  tm.tm_isdst  = -1;
  clock = mktime(&tm);
  tp->tv_sec = clock;
  tp->tv_usec = wtm.wMilliseconds * 1000;
  return 0;
}
#endif
double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);

}

void initialDataInt(int *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (int)(rand()&0xff);
        // ip[i] = i;
    }  
}

void initialData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
        // ip[i] = i;
    }
}

void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp,dev));
  printf("Using device %d: %s\n",dev,deviceProp.name);
  CHECK(hipSetDevice(dev));

}

void checkRes(float *hostRef, float *gpuRef, int size)
{
    int flag = 0;
    double epsilon = 1.0E-4;
    for (int i = 0; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            printf("Results don\'t match!\n");
            printf("%f(hostRef[%d] )!= %f(gpuRef[%d])\n",hostRef[i],i,gpuRef[i],i);
            flag = 1;
            break;
        }
    }
    if (!flag)
    {
        printf("Check result success! All values are identical !\n");
    }
}

#endif