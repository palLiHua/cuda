#include "hip/hip_runtime.h"
#include "../include/util.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

int threads_per_block = 256;
int blocks_per_grid = 30;

__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState *state = states + tid;

    hiprand_init(9384, tid, 0, state);

    for (i = tid; i < N; i += nthreads)
    {
        float rand = hiprand_uniform(state);
        rand = rand * 2;
        out[i] = rand;
    }
}

__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < N; i += nthreads)
    {
        float rand = randomValues[i];
        rand = rand * 2;
        out[i] = rand;
    }
}

void use_host_api(int N)
{
    hiprandGenerator_t randGen;
    CHECK_CURAND(hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT));

    float *dRand, *dOut, *hOut;
    CHECK(hipMalloc((float**)&dRand, sizeof(float) * N));
    CHECK(hipMalloc((float**)&dOut, sizeof(float) * N));

    hOut = (float*)malloc(sizeof(float) * N);

    CHECK_CURAND(hiprandGenerateUniform(randGen, dRand, N));

    host_api_kernel<<<blocks_per_grid, threads_per_block>>>(dRand, dOut, N);

    CHECK(hipMemcpy(hOut, dOut, sizeof(float) * N, hipMemcpyDeviceToHost));

    printf("Sampling of output from host API:\n");

    for (int i = 0; i < 10; i++)
    {
        printf("%2.4f\n", hOut[i]);
    }

    printf("...\n");

    free(hOut);
    CHECK(hipFree(dRand));
    CHECK(hipFree(dOut));
    CHECK_CURAND(hiprandDestroyGenerator(randGen));
}

void use_device_api(int N)
{
    static hiprandState *states = NULL;
    CHECK(hipMalloc((void **)&states, sizeof(hiprandState) *
                threads_per_block * blocks_per_grid));
    
    float *dOut, *hOut;

    CHECK(hipMalloc((void **)&dOut, sizeof(float) * N));
    hOut = (float *)malloc(sizeof(float) * N);

    device_api_kernel<<<blocks_per_grid, threads_per_block>>>(states, dOut, N);

    CHECK(hipMemcpy(hOut, dOut, sizeof(float) * N, hipMemcpyDeviceToHost));
    
    printf("Sampling of output from device API:\n");

    for (int i = 0; i < 10; i++)
    {
        printf("%2.4f\n", hOut[i]);
    }

    printf("...\n");

    free(hOut);
    CHECK(hipFree(dOut));
    CHECK(hipFree(states));

}

int main(int argc, char **argv)
{
    int N = 8388608;

    use_host_api(N);
    use_device_api(N);

    return 0;
}