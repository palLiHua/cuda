#include "hip/hip_runtime.h"
#include "../include/util.h"
#include <hipsparse.h> 

int rows = 4;
int cols = 4;
int check_rows = 4;

void check_res(float *A, float *X, float *Y)
{
    int flag = 0;
    for (int i = 0; i < check_rows; i++)
    {
        float tmp = 0.0;
        for (int j = 0; j < cols; j++)
        {
            tmp += A[i*cols + j] * X[j];
        }
        if (abs(tmp - Y[i]) > 0.0001)
        {
            printf("%dth element doesn't match, it should be %f, but get %f\n", i, tmp, Y[i]);
            flag = 1;
            break;
        }
    }
    if (!flag) printf("Check %d elements successfully\n", check_rows);
}

void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

int generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            float *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0f;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0f)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

int main(int argc, char **argv)
{
    srand(9384);

    hipsparseHandle_t handle = 0;

    float* A;
    float* X;
    float* Y;

    float *dA, *dX, *dY;
    
    int trueNnz = generate_random_dense_matrix(rows, cols, &A); // M * N
    generate_random_vector(cols, &X);   // N * 1
    generate_random_vector(rows, &Y);   // M * 1

    CHECK_CUSPARSE(hipsparseCreate(&handle));

    CHECK(hipMalloc((float**)&dA, sizeof(float) * rows * cols));
    CHECK(hipMalloc((float**)&dX, sizeof(float) * cols));
    CHECK(hipMalloc((float**)&dY, sizeof(float) * rows));

    CHECK(hipMemcpy(dA, A, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dX, X, sizeof(float) * cols, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dY, Y, sizeof(float) * rows, hipMemcpyHostToDevice));

    /**
     * convert data format
     */
    // hipsparseMatDescr_t descrA;
    hipsparseDnMatDescr_t descrA;
    // int *dNnzPerRow;
    // int totalNnz;
    // CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&descrA, rows, cols, cols, dA, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, rows, cols, descrA, dA,
    //                             rows, dNnzPerRow, &totalNnz)); // count the number of non-zero

    // if (totalNnz != trueNnz)
    // {
    //     fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
    //             "value: expected %d but got %d\n", trueNnz, totalNnz);
    //     return 1;
    // }

    hipsparseSpMatDescr_t descrB;
    int *d_csr_offset;
    size_t bufferSize = 0;
    CHECK(hipMalloc((int**)&d_csr_offset, sizeof(int) * (rows + 1)));
    CHECK_CUSPARSE(hipsparseCreateCsr(&descrB, rows, cols, 0, d_csr_offset, 
                                    NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                    handle, descrA, descrB,
                                    HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                    &bufferSize) );
    void *dBuffer = NULL;
    CHECK( hipMalloc(&dBuffer, bufferSize));

    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, descrA, descrB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );
    // int64_t num_rows_tmp, num_cols_tmp, nnz;
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(descrB, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) );
    if (nnz != trueNnz)
    {
        printf("Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %ld\n", trueNnz, nnz);
        return 1;
    }
    float *d_csr_values;
    int *d_csr_columns;
    CHECK( hipMalloc((void**) &d_csr_columns, nnz * sizeof(int))   );
    CHECK( hipMalloc((void**) &d_csr_values,  nnz * sizeof(float)) );

    CHECK_CUSPARSE( hipsparseCsrSetPointers(descrB, d_csr_offset, d_csr_columns,
                                           d_csr_values) );
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, descrA, descrB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) );

    /**
     * check convert result
     */
    int *offset = (int*)malloc(sizeof(int) * (rows + 1));
    int *idx = (int*)malloc(sizeof(int) * nnz);
    float *values = (float*)malloc(sizeof(float) * nnz);
    
    CHECK(hipMemcpy(offset, d_csr_offset, sizeof(int) * (rows + 1), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(idx, d_csr_columns, sizeof(int) * nnz, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(values, d_csr_values, sizeof(float) * nnz, hipMemcpyDeviceToHost));

    for (int i = 0; i < nnz; i++)
    {
        printf("i = %d, idx = %d, values = %f\n", i, idx[i], values[i]);
    }
    for (int i = 0; i < rows + 1; i++)
    {
        printf("rows = %d, offset = %d\n", i, offset[i]);
    }
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%f ", A[i*cols + j]);
        }
        printf("%f ", X[i]);
        printf("\n");
    }
    free(idx);
    free(offset);
    free(values);
    /**
     * A * X
     */
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseDnVecDescr_t vecX, vecY;
    size_t bufferSize_ = 0;
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, cols, dX, HIP_R_32F) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, rows, dY, HIP_R_32F) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, descrB, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize_) );
    void *dBuffer_ = NULL;
    CHECK( hipMalloc(&dBuffer_, bufferSize_) );
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, descrB, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) );

    /**
     * copy back result
     */

    CHECK(hipMemcpy(Y, dY, sizeof(float) * rows, hipMemcpyDeviceToHost));
    
    // for (int i = 0; i < rows; i++)
    // {
    //     printf("%2.2f\n", Y[i]);
    // }
    check_res(A, X, Y);


    CHECK(hipFree(dA));
    CHECK(hipFree(dX));
    CHECK(hipFree(dY));
    CHECK(hipFree(dBuffer));
    CHECK(hipFree(dBuffer_));
    CHECK(hipFree(d_csr_offset));
    CHECK(hipFree(d_csr_values));
    CHECK(hipFree(d_csr_columns));

    // CHECK(hipFree(dNnzPerRow));

    free(A);
    free(X);
    free(Y);

    CHECK_CUSPARSE( hipsparseDestroyDnMat(descrA) );
    CHECK_CUSPARSE( hipsparseDestroySpMat(descrB) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseDestroy(handle) );

    return 0;
}